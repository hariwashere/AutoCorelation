#include "hip/hip_runtime.h"
__global__ void compute_Dontcares(char* image,int* row_flag,int* col_flag,char* result, int i, int j, int image_width)
{
	int row = (blockIdx.x + i);
	int col = threadIdx.x + j;
	if(image[row*image_width+col] == image[(row-i)*image_width+col-j])
	{
		if(image[row*image_width+col] == '1')
			result[(row-i)*(image_width-j)+ (col-j)]='1';

		else//(image[row*image_width+col] == '0')
			result[(row-i)*(image_width-j)+ (col-j)]='0';

		row_flag[row-i]=1;
		col_flag[col-j]=1;
	}
	else
		result[(row-i)*(image_width-j)+ (col-j)]='o';
	//result++;
}


__global__ void compute_Result(char* ptr_ptr ,int image_height, int image_width, int row_start,int col_start, int col_end, char* result2_ptr, int j)    {
	int row = blockIdx.x + row_start;
	int col = threadIdx.x + col_start;
	result2_ptr[(row - row_start) * (col_end - col_start + 1) + (col - col_start)] = ptr_ptr[row*(image_width-j) + col];
}



ConsensusGrid consensus_parallel(int i, int j,char* image, int image_height, int image_width)
{
	ConsensusGrid consensus_grid;
	char* image_ptr;
	int* row_flag_ptr;
	int*  col_flag_ptr;
	char * ptr_ptr;
	char*  result2_ptr;
	char* result_ptr;

	char *result = (char*)malloc(sizeof(char)*(image_height-i)*(image_width-j));
	char* ptr = result;

	int* row_flag=(int*)calloc(sizeof(int),(image_height-i));
	int* col_flag=(int*)calloc(sizeof(int),(image_width-j));
	//copy result and image to gpu
	//compute the result array
	//copy result back to cpu
	hipMalloc((void**)&image_ptr, sizeof(char)*(image_height*image_width));
	hipMemcpy(image_ptr, image, sizeof(char)*(image_height*image_width) ,  hipMemcpyHostToDevice);
	hipMalloc((void**)&row_flag_ptr, sizeof(int)*(image_height-i));
	hipMemcpy(row_flag_ptr, row_flag, sizeof(int)*(image_height-i) ,  hipMemcpyHostToDevice);
	hipMalloc((void**)&col_flag_ptr, sizeof(int)*(image_width-j));
	hipMemcpy(col_flag_ptr, col_flag, sizeof(int)*(image_width-j) ,  hipMemcpyHostToDevice);
	hipMalloc((void**)&result_ptr, sizeof(char)*(image_width-j)*(image_height-i));


	// __global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter);
	compute_Dontcares<<<(image_height-i),(image_width-j)>>>(image_ptr,row_flag_ptr, col_flag_ptr, result_ptr,i,j, image_width);

	hipDeviceSynchronize();
	hipMemcpy(result, result_ptr, sizeof(char)*(image_height-i)*(image_width-j),  hipMemcpyDeviceToHost);
	hipMemcpy(row_flag, row_flag_ptr, sizeof(int)*(image_height-i),  hipMemcpyDeviceToHost);
	hipMemcpy(col_flag, col_flag_ptr, sizeof(int)*(image_width-j),  hipMemcpyDeviceToHost);
	hipMemcpy(image, image_ptr, sizeof(char)*(image_height)*(image_width),  hipMemcpyDeviceToHost);
	//////Printing
	//if(i ==0 &&j == 0)
/*	cout << "i = " << i << " j = " << j << endl;
	{

		/*for(int ii=0;ii<image_height-i;ii++)
		{
			cout << "rf " << row_flag[ii] << endl;
		}*/
/*		for(int ii=0; ii<(image_height-i)*(image_width-j); ii++)
		{
			cout <<"p " << ptr[ii] << ", " ;
		}
	}
	cout<< endl << endl;
*/	int row_start,row_end;
	int col_start,col_end;
	for(row_start=0; row_start<(image_height-i); row_start++)
	{
		if(row_flag[row_start]==1)
			break;
	}
	for(row_end=(image_height-i-1); row_end>=0; row_end--)
	{
		if(row_flag[row_end]==1)
			break;
	}
	for(col_start=0; col_start<(image_width-j); col_start++)
	{
		if(col_flag[col_start]==1)
			break;
	}
	for(col_end=(image_width-j-1); col_end>=0; col_end--)
	{
		if(col_flag[col_end]==1)
			break;
	}

	if((row_start > row_end) || (col_start > col_end) )
	{
		consensus_grid.result = NULL;
		consensus_grid.height = 0;
		consensus_grid.width = 0;
		return consensus_grid;
	}

	char* result2 = (char*)malloc(sizeof(char)*(row_end-row_start+1)*(col_end-col_start+1));
	hipMalloc((void**)&result2_ptr, sizeof(char)*(row_end-row_start+1)*(col_end-col_start+1));
	//hipMemcpy(col_flag_ptr, col_flag, sizeof(image_width-j) ,  hipMemcpyHostToDevice);
	hipMalloc((void**)&ptr_ptr, sizeof(char)*(image_height-i)*(image_width-j));
	hipMemcpy(ptr_ptr, ptr, sizeof(char)*(image_height-i)*(image_width-j) ,  hipMemcpyHostToDevice);

	consensus_grid.result = result2;

	compute_Result<<<(row_end-row_start+1),(col_end-col_start+1)>>>(ptr_ptr , image_height, image_width, row_start,col_start, col_end, result2_ptr, j);

	hipDeviceSynchronize();
	hipMemcpy(result2, result2_ptr, sizeof(char)*(row_end-row_start+1)*(col_end-col_start+1),  hipMemcpyDeviceToHost);
	 

	consensus_grid.height = row_end - row_start + 1;
	consensus_grid.width = col_end - col_start + 1;
	consensus_grid.occurance = NULL;
	consensus_grid.occurance_count = 0;

	free(ptr);
	free(row_flag);
	free(col_flag);
	return consensus_grid;
}
