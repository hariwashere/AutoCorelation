#include "hip/hip_runtime.h"
__global__ compute_Dontcares(char* image_ptr,char* row_flag_ptr,char* col_flag_ptr,char* result_ptr, int i, int j)
    {
        int row = blockIdy.y*blockDim.y + threadIdx.y + i;
        int col = blockIdx.x*blockDim.x + threadIdx.x + j;
        if(image[row*image_width+col] == image[(row-i)*image_width+col-j])
        {
            if(image[row*image_width+col] == '1')
                result[(row-i)*(image_width-j)+ (col-j)]='1';

            else if(image[row*image_width+col] == '0')
                result[(row-i)*(image_width-j)+ (col-j)]='0';

            row_flag[row-i]=1;
            col_flag[col-j]=1;
        }
        else
            result[(row-i)*(image_width-j)+ (col-j)]='o';
        //result++;
    }


__global__ compute_Result(char* ptr_ptr ,int image_height, int image_width, int row_start,int col_start)
    {
        int row = blockIdy.y*blockDim.y + threadIdx.y + row_start;
        int col = blockIdx.x*blockDim.x + threadIdx.x + col_start;
        result2_ptr[(row - row_start) * (col_end - col_start + 1) + (col - col_start)] = ptr[row*(image_width-j) + col];
    }



ConsensusGrid consensus_parallel(int i, int j,char* image, int image_height, int image_width)
{
    ConsensusGrid consensus_grid;
    char *result = (char*)malloc(sizeof(char)*(image_height-i)*(image_width-j));
    char* ptr = result;

    int* row_flag=(int*)calloc(sizeof(int),(image_height-i));
    int* col_flag=(int*)calloc(sizeof(int),(image_width-j));
    //copy result and image to gpu
//compute the result array
//copy result back to cpu
    hipMalloc((void**)&image_ptr, sizeof(image_height*image_width));
    hipMemcpy(image_ptr, image, sizeof(image_height*image_width) ,  hipMemcpyHostToDevice);
    hipMalloc((void**)&row_flag_ptr, sizeof(image_height-i));
    hipMemcpy(row_flag_ptr, row_flag, sizeof(image_height-i) ,  hipMemcpyHostToDevice);
    hipMalloc((void**)&col_flag_ptr, sizeof(image_width-j));
    hipMemcpy(col_flag_ptr, col_flag, sizeof(image_width-j) ,  hipMemcpyHostToDevice);

   // __global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter);
    compute_Dontcares<<<(image_height-i),(image_width-j)>>>(image_ptr,row_flag_ptr, col_flag_ptr, result_ptr,i,j);

    hipDeviceSynchronize();
    hipMemcpy(result, result_ptr, sizeof((image_height-i)*(image_width-j)),  hipMemcpyDeviceToHost);
    hipMemcpy(row_flag, row_flag_ptr, sizeof((image_height-i)),  hipMemcpyDeviceToHost);
    hipMemcpy(col_flag, col_flag_ptr, sizeof((image_width-j)),  hipMemcpyDeviceToHost);

    int row_start,row_end;
    int col_start,col_end;
    for(row_start=0; row_start<(image_height-i); row_start++)
    {
        if(row_flag[row_start]==1)
            break;
    }
    for(row_end=(image_height-i-1); row_end>=0; row_end--)
    {
        if(row_flag[row_end]==1)
            break;
    }
    for(col_start=0; col_start<(image_width-j); col_start++)
    {
        if(col_flag[col_start]==1)
            break;
    }
    for(col_end=(image_width-j-1); col_end>=0; col_end--)
    {
        if(col_flag[col_end]==1)
            break;
    }

    if((row_start > row_end) || (col_start > col_end) )
    {
        consensus_grid.result = NULL;
        consensus_grid.height = 0;
        consensus_grid.width = 0;
        return consensus_grid;
    }

    char *result2 = (char*)malloc(sizeof(char)*(row_end-row_start+1)*(col_end-col_start+1));
    hipMalloc((void**)&result2_ptr, sizeof((char)*(row_end-row_start+1)*(col_end-col_start+1)));
    //hipMemcpy(col_flag_ptr, col_flag, sizeof(image_width-j) ,  hipMemcpyHostToDevice);
    hipMalloc((void**)&ptr_ptr, sizeof((image_height-i)*(image_width-j));
    hipMemcpy(ptr_ptr, ptr, sizeof((image_height-i)*(image_width-j)) ,  hipMemcpyHostToDevice);

    consensus_grid.result = result2;

    compute_Result<<<(row_end-row_start),(col_end-col_start)>>>(ptr_ptr , image_height, image_width, row_start,col_start);

    hipDeviceSynchronize();
    hipMemcpy(result2, result2_ptr, sizeof((char)*(row_end-row_start+1)*(col_end-col_start+1)),  hipMemcpyDeviceToHost);

    consensus_grid.height = row_end - row_start + 1;
    consensus_grid.width = col_end - col_start + 1;
    consensus_grid.occurance = NULL;
    consensus_grid.occurance_count = 0;

    free(ptr);
    free(row_flag);
    free(col_flag);
    return consensus_grid;
}
